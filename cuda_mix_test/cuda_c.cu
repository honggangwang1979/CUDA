#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include <time.h>

__global__ void testkernel(int *data, int size){

  for (int i = 1; i < size; i++) data[0] += data[i];
}

extern "C" {
 int cudatestfunc(int *data, int size){
  int *d_data;
  //cudaMalloc((void **) &d_data, size*sizeof(int));
  std::cout<<hipGetErrorName(hipMalloc( (void **)&d_data, size * sizeof(int) ))<<std::endl;
  std::cout<<hipGetErrorName(hipMemcpy(d_data, data, size*sizeof(int), hipMemcpyHostToDevice)) << std::endl;
  testkernel<<<1,1>>>(d_data, size);
  int result;
  std::cout<<hipGetErrorName(hipMemcpy(&result, d_data, sizeof(int), hipMemcpyDeviceToHost)) << std::endl;
  std::cout<<hipGetErrorName(hipMemcpy(data, d_data, size*sizeof(int), hipMemcpyDeviceToHost)) << std::endl;
  return result;
 }
}
