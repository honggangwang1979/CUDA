#include <iostream>
#include <stdio.h> 
#include <stdlib.h> 
#include <string.h>
#include <hip/hip_runtime.h> 
 
#include <time.h>

// simple kernel function that adds two vectors 
__global__ void vect_add(float *a, float *b, int N) { 
	int idx = threadIdx.x; if (idx<N) a[idx] = a[idx] + b[idx]; 
} 

__global__ void cube_add(double *a, double *b, double *c, int N_i, int N_j, int N_k) { 
	int index_i = blockIdx.x*blockDim.x + threadIdx.x; 
	int index_j = blockIdx.y*blockDim.y + threadIdx.y;
	int index_k = blockIdx.z*blockDim.z + threadIdx.z;

	int strid_i = blockDim.x * gridDim.x;
	int strid_j = blockDim.y * gridDim.y;
	int strid_k = blockDim.z * gridDim.z;
	//int total_does = 0;


        for ( int i = index_i; i<N_i; i+=strid_i ){
        	for ( int j = index_j; j<N_j; j+=strid_j ){
        		for ( int k = index_k; k<N_k; k+=strid_k ){
				c[i*N_j*N_k+j*N_k+k] = a[i*N_j*N_k+j*N_k+k] + b[i*N_j*N_k+j*N_k+k];
				//c[i+j*N_i+k*N_i*N_j] = a[i+j*N_i+k*N_i*N_j] + b[i+j*N_i+k*N_i*N_j];
				//total_does++;
			}
		}
	}

	//printf("one thread runs %d times. \n", total_does);
} 
__global__ void cube_minus(double *a, double *b, double *c, int N_i, int N_j, int N_k) { 
	int index_i = blockIdx.x*blockDim.x + threadIdx.x; 
	int index_j = blockIdx.y*blockDim.y + threadIdx.y;
	int index_k = blockIdx.z*blockDim.z + threadIdx.z;

	int strid_i = blockDim.x * gridDim.x;
	int strid_j = blockDim.y * gridDim.y;
	int strid_k = blockDim.z * gridDim.z;
	int total_does = 0;


        for ( int i = index_i; i<N_i; i+=strid_i ){
        	for ( int j = index_j; j<N_j; j+=strid_j ){
        		for ( int k = index_k; k<N_k; k+=strid_k ){
				c[i*N_j*N_k+j*N_k+k] = a[i*N_j*N_k+j*N_k+k] - b[i*N_j*N_k+j*N_k+k];
				//c[i+j*N_i+k*N_i*N_j] = a[i+j*N_i+k*N_i*N_j] + b[i+j*N_i+k*N_i*N_j];
				total_does++;
			}
		}
	}

	printf("one thread runs %d times. \n", total_does);
}

__global__ void cube_product(double *a, double *b, double *c, int N_i, int N_j, int N_k) { 
	int index_i = blockIdx.x*blockDim.x + threadIdx.x; 
	int index_j = blockIdx.y*blockDim.y + threadIdx.y;
	int index_k = blockIdx.z*blockDim.z + threadIdx.z;

	int strid_i = blockDim.x * gridDim.x;
	int strid_j = blockDim.y * gridDim.y;
	int strid_k = blockDim.z * gridDim.z;
	//int total_does = 0;


        for ( int i = index_i; i<N_i; i+=strid_i ){
        	for ( int j = index_j; j<N_j; j+=strid_j ){
        		for ( int k = index_k; k<N_k; k+=strid_k ){
				for( int m=0; m<1000; m++ )
					c[i*N_j*N_k+j*N_k+k] = a[i*N_j*N_k+j*N_k+k] * b[i*N_j*N_k+j*N_k+k];
				//c[i+j*N_i+k*N_i*N_j] = a[i+j*N_i+k*N_i*N_j] + b[i+j*N_i+k*N_i*N_j];
				//total_does++;
			}
		}
	}

	//printf("one thread runs %d times. \n", total_does);
}

// function called from main fortran program extern "C" 
// this following line works for the main() commented by the end of this code
//extern "C" void kw_simple_3d_opr_(double *a, double *b, double *c, int *pN_i, int *pN_j, int *pN_k, char *pOperator) 
void kernel_wrapper_(double *a, double *b, double *c, int *pN_i, int *pN_j, int *pN_k, char *pOperator) 
{ 

	clock_t start = clock();
	double *a_d, *b_d, *c_d; 
	// declare GPU vector copies 
	int blocks = 1; 
	int blockDim_x=16, blockDim_y=16, blockDim_z=4;
	int gridDim_x, gridDim_y, gridDim_z;

	int N_i = *pN_i; 
	int N_j = *pN_j; 
	int N_k = *pN_k; 
	char Operator = *pOperator;
        
        gridDim_x = N_i%blockDim_x == 0 ? N_i/blockDim_x: N_i/blockDim_x +1;
        gridDim_y = N_j%blockDim_y == 0 ? N_j/blockDim_y: N_j/blockDim_y +1;
        gridDim_z = N_k%blockDim_z == 0 ? N_k/blockDim_z: N_k/blockDim_z +1;

	//dimention cannot be zero,otherwise wierd things may happen
	//gridDim_x = 0;  
	//gridDim_y = 0;

	dim3 grid(gridDim_x, gridDim_y, gridDim_z);
	dim3 block(blockDim_x, blockDim_y, blockDim_z);


	// i*j*k threads offloaded on GPU // Allocate memory on GPU 
	int Tsize = N_i *N_j *N_k *sizeof(double);
	//std::cout<<cudaGetErrorName(cudaMalloc( (void **)&a_d, Tsize))<<std::endl;
	//std::cout<<cudaGetErrorName(cudaMalloc( (void **)&b_d, Tsize))<<std::endl;
	//std::cout<<cudaGetErrorName(cudaMalloc( (void **)&c_d, Tsize))<<std::endl;

	// CudaMallocHost use pinned host memory which is supposed to be faster than CudaMalloc()
	//std::cout<<cudaGetErrorName(cudaMallocHost( (void **)&a_d, Tsize))<<std::endl;
	//std::cout<<cudaGetErrorName(cudaMallocHost( (void **)&b_d, Tsize))<<std::endl;
	//std::cout<<cudaGetErrorName(cudaMallocHost( (void **)&c_d, Tsize))<<std::endl;

	// CudaHostAlloc use zero-copy memory which is supposed to be even faster than CudaMallocHost()
	//std::cout<<cudaGetErrorName(cudaHostAlloc( (void **)&a_d, Tsize,cudaHostAllocMapped))<<std::endl;
	//std::cout<<cudaGetErrorName(cudaHostAlloc( (void **)&b_d, Tsize,cudaHostAllocMapped))<<std::endl;
	//std::cout<<cudaGetErrorName(cudaHostAlloc( (void **)&c_d, Tsize,cudaHostAllocMapped))<<std::endl;

	// copy vectors from CPU to GPU 
	//std::cout<<cudaGetErrorName(cudaMemcpy( a_d, a, Tsize, cudaMemcpyHostToDevice ))<<std::endl; 
	//std::cout<<cudaGetErrorName(cudaMemcpy( b_d, b, Tsize, cudaMemcpyHostToDevice ))<<std::endl; 
	//std::cout<<cudaGetErrorName(cudaMemcpy( c_d, c, Tsize, cudaMemcpyHostToDevice ))<<std::endl; 

	printf("Time elapsed: %f\n", ((double)clock() - start) / CLOCKS_PER_SEC);

	printf("in kernal before cube_add\n");

	// call function on GPU
       	//vect_add<<< blocks, N >>>( a_d, b_d, N); 
	switch ( Operator ){
		case '+':
        		//cube_add<<<grid, block>>>( a_d, b_d, c_d, N_i, N_j, N_k); 
        		cube_add<<<grid, block>>>( a, b, c, N_i, N_j, N_k); 
			break;
		case '-':
        		//cube_minus<<<grid, block>>>( a_d, b_d, c_d, N_i, N_j, N_k); 
			break;
		case '*':
        		//cube_product<<<grid, block>>>( a_d, b_d, c_d, N_i, N_j, N_k); 
			break;
	}
	//cudaDeviceSynchronize();
	// copy vectors back from GPU to CPU 

	printf("in kernal after cube_add\n");

	//cudaMemcpy( a, a_d, Tsize, cudaMemcpyDeviceToHost ); 
	//cudaMemcpy( b, b_d, Tsize, cudaMemcpyDeviceToHost ); 
	//cudaMemcpy( c, c_d, Tsize, cudaMemcpyDeviceToHost ); 

       	// free GPU memory 
	//cudaFree(a_d);
        //cudaFree(b_d); 
        //cudaFree(c_d); 
	return; 
} 

int main()
{
	int N_i = 2, N_j=3, N_k=4;
	double a[N_i][N_j][N_k];
	double b[N_i][N_j][N_k];
	double c[N_i][N_j][N_k];

	double *pa, *pb, *pc;

	int Tsize = N_i * N_j *N_k * sizeof(double);

	hipMallocManaged(&pa, Tsize );
	hipMallocManaged(&pb, Tsize );
	hipMallocManaged(&pc, Tsize );

	memset(pa, 0x0, Tsize);
	memset(pb, 0x0, Tsize);
	memset(pc, 0x0, Tsize);

	for ( int k=0; k<N_k; k++){
		for ( int j=0; j<N_j; j++){
			for ( int i=0; i<N_i; i++) {
				//a[i][j][k] = i + j*N_i + k * N_j * N_i;
				//b[i][j][k] = a[i][j][k];
				//c[i][j][k] = 0.0;
				pa[i+j*N_i+k*N_i*N_j] = i + j*N_i + k *N_j * N_i;
				pb[i+j*N_i+k*N_i*N_j] = i + j*N_i + k *N_j * N_i;
				pc[i+j*N_i+k*N_i*N_j] = 0;
				printf("pa[%d][%d][%d]=%f\n",i,j,k,pa[i+j*N_i+k*N_i*N_j]);
			}
		}
	}


	printf("Tsize = %d, sizeof(a) =%d \n", Tsize, sizeof(a) );

        //pa = (double*)calloc(N_i * N_j * N_k, sizeof(double));
        //pb = (double*)calloc(N_i * N_j * N_k, sizeof(double));
        //pc = (double*)calloc(N_i * N_j * N_k, sizeof(double));



	//memcpy(pa, a, Tsize);
	//memcpy(pb, b, Tsize);
	//memcpy(pc, c, Tsize);

	char Operator = '+';
	clock_t start = clock();
	kernel_wrapper_((double *)pa,(double *)pb,(double *)pc, &N_i, &N_j, &N_k, &Operator);
	printf("Time elapsed: %f\n", ((double)clock() - start) / CLOCKS_PER_SEC);

	for ( int i=0; i<N_i; i++) {
		for ( int j=0; j<N_j; j++){
			for ( int k=0; k<N_k; k++){
				printf("c[%d][%d][%d]=%f\n",i,j,k,pc[i+j*N_i+k*N_j*N_i]);
			}
		}
	}

	hipFree(pa);
	hipFree(pb);
	hipFree(pc);
	return 0;
} 
