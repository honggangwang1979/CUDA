// this code is to test the Unified Memeory vs non-unified memory

#include <hipfft/hipfft.h>
#include <iostream>
#include <string>

//using namespace std;
const int dataSize  = 1048576*32;

void setupWave(const int ds, hipfftComplex *d){
  for (int i = 0; i < ds; i++){
    d[i].x = 1.0f;
    d[i].y = 0.0f;}
}

int main(){

hipfftComplex *inData, *outData;

hipfftHandle plan;
hipfftPlan1d(&plan, dataSize, HIPFFT_C2C, 1);

hipMallocManaged(&inData, dataSize * sizeof(hipfftComplex));
hipMallocManaged(&outData, dataSize * sizeof(hipfftComplex));

hipEvent_t start_before_memHtoD, start_kernel, stop_kernel,
                stop_after_memDtoH;
hipEventCreate(&start_kernel);
hipEventCreate(&start_before_memHtoD);
hipEventCreate(&stop_kernel);
hipEventCreate(&stop_after_memDtoH);

setupWave(dataSize, inData);

hipEventRecord(start_before_memHtoD);
hipMemPrefetchAsync(inData, dataSize * sizeof(hipfftComplex), 0);
hipMemPrefetchAsync(outData, dataSize * sizeof(hipfftComplex), 0);
hipDeviceSynchronize();

hipEventRecord(start_kernel);

//call kernal 
hipfftExecC2C(plan, inData, outData, HIPFFT_FORWARD);

hipEventRecord(stop_kernel);

hipEventSynchronize(stop_kernel);

float sum = 0;
for (int i = 0; i < dataSize; i++) {
        sum += outData[i].x + outData[i].y;
}
hipEventRecord(stop_after_memDtoH);
hipEventSynchronize(stop_after_memDtoH);

std::cout << "sum for UM is " << sum << std::endl;


float umTime = 0;
float overallUmTime = 0;
hipEventElapsedTime(&umTime, start_kernel, stop_kernel);
hipEventElapsedTime(&overallUmTime, start_before_memHtoD,
                stop_after_memDtoH);

std::string resultString_um = std::to_string(dataSize) + " samples took " + std::to_string(umTime) + "ms,  Overall: " + std::to_string(overallUmTime) + "\n";

std::cout << resultString_um;
hipEventDestroy(start_kernel);
hipEventDestroy(stop_kernel);
hipFree(inData);
hipFree(outData);
hipEventDestroy(start_before_memHtoD);
hipEventDestroy(stop_after_memDtoH);

hipfftDestroy(plan);


// start non UM process:

hipfftComplex *d_inData;
hipfftComplex *d_outData;
inData = (hipfftComplex*) (malloc(sizeof(hipfftComplex) * dataSize));
outData = (hipfftComplex*) (malloc(sizeof(hipfftComplex) * dataSize));
hipMalloc((void**) (&d_inData), dataSize * sizeof(hipfftComplex));
hipMalloc((void**) (&d_outData), dataSize * sizeof(hipfftComplex));
//hipfftHandle plan;
hipfftPlan1d(&plan, dataSize, HIPFFT_C2C, 1);

//hipEvent_t start_before_memHtoD, start_kernel, stop_kernel,
//                stop_after_memDtoH;
hipEventCreate(&start_kernel);
hipEventCreate(&start_before_memHtoD);
hipEventCreate(&stop_kernel);
hipEventCreate(&stop_after_memDtoH);

setupWave(dataSize, inData);

hipEventRecord(start_before_memHtoD);
hipMemcpy(d_inData, inData, dataSize * sizeof(hipfftComplex),
                                        hipMemcpyHostToDevice);
hipEventRecord(start_kernel);

hipfftExecC2C(plan, d_inData, d_outData, HIPFFT_FORWARD);

hipEventRecord(stop_kernel);

hipEventSynchronize(stop_kernel);

hipMemcpy(outData, d_outData, dataSize * sizeof(hipfftComplex),
                hipMemcpyDefault);

 sum = 0;
for (int i = 0; i < dataSize; i++) {
        sum += outData[i].x + outData[i].y;
}
hipEventRecord(stop_after_memDtoH);
hipEventSynchronize(stop_after_memDtoH);

std::cout << "sum for non-UM is " << sum << std::endl;

//float umTime = 0;
//float overallUmTime = 0;
hipEventElapsedTime(&umTime, start_kernel, stop_kernel);
hipEventElapsedTime(&overallUmTime, start_before_memHtoD,
                stop_after_memDtoH);

resultString_um = std::to_string(dataSize) + " samples took "
                + std::to_string(umTime) + "ms,  Overall: "
                + std::to_string(overallUmTime) + "\n";
std::cout << resultString_um;
free(outData);
free(inData);
hipFree(d_outData);
hipFree(d_inData);
hipEventDestroy(start_kernel);
hipEventDestroy(stop_kernel);

hipEventDestroy(start_before_memHtoD);
hipEventDestroy(stop_after_memDtoH);

hipfftDestroy(plan);

}

